#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

#include <cub/block/block_load.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/block/block_reduce.cuh>

#define DEBUG_LEVEL 0
#include "bfs_hops.cuh"

inline void calculate_kernel_config(int thread_num, int& block_size, int& grid_size){
    //int threadsPerBlock_up = ((thread_num + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
    //block_size = thread_num > BLOCK_MAX_SIZE ? BLOCK_MAX_SIZE : threadsPerBlock_up;
    block_size = BLOCK_MAX_SIZE;
    grid_size = (thread_num + BLOCK_MAX_SIZE - 1) / BLOCK_MAX_SIZE;
}

__device__ int update_frontiers_num;

__global__ void initial_output_bitmap_o2(int* output_bitmap, int* output_num, int bitmap_len){
    int glb_tid = blockIdx.x * blockDim.x + threadIdx.x;
    CUDA_DEBUG(glb_tid, "Initial Output Bitmap...\n");
    int block_size = blockDim.x;
    int grid_size = gridDim.x;
    int threadsPerGrid = block_size * grid_size;

    for (int i = glb_tid; i < bitmap_len; i += threadsPerGrid) {
        output_bitmap[i] = 0;
    }
    if(glb_tid == 0) *output_num = 0;
}

__global__ void get_output_frontiers_o2(int* g_offset, int* g_edges, int node_num, int edge_num,
                                    int* input_bitmap, int nf_num,
                                    int* output_bitmap, int* output_num,
                                    int* status_bitmap, int bitmap_len,
                                    int* update_frontiers){
    int glb_tid = blockIdx.x * blockDim.x + threadIdx.x;
    CUDA_DEBUG(glb_tid, "Get Output Frontiers O1...\n");
    int blk_tid = threadIdx.x;
    int block_size = blockDim.x;
    int grid_size = gridDim.x;
    int bid = blockIdx.x;

    typedef hipcub::BlockReduce<int, BLOCK_MAX_SIZE> BlockReduce;
    typedef hipcub::BlockScan<int, BLOCK_MAX_SIZE> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    __shared__ typename BlockReduce::TempStorage temp_storage2;
    __shared__ int blk_output_num;

    //Todo: sparse store
    extern __shared__ int dynamic_smem[];
    int* blk_status_bitmap = dynamic_smem;
    int* blk_output_bitmap = dynamic_smem + bitmap_len;
    int* blk_input_frontiers = dynamic_smem + 2 * bitmap_len;

    // 1. load bitmap into shared memory (Todo:async)
    for (int i = blk_tid; i < bitmap_len; i += block_size){
        blk_status_bitmap[i] = status_bitmap[i];
        blk_output_bitmap[i] = 0;
    }
    if(blk_tid == 0) blk_output_num = 0;
    //__syncthreads();

    // 2. get input frontiers
    int bitmap;
    int v_num[1];
    int v_offset[1];
    int total_v_num_per_it;

    int input_bitmap_remaind = bitmap_len % grid_size;
    int input_bitmap_num = bitmap_len / grid_size;
    int input_bitmap_offset = bid * input_bitmap_num;
    // assign bitmap
    if (bid < input_bitmap_remaind) {
        input_bitmap_num += 1;
        input_bitmap_offset += bid;
    } else {
        input_bitmap_offset += input_bitmap_remaind;
    }

    int it = input_bitmap_num / block_size;
    int blk_input_bitmap_remaind = input_bitmap_num % block_size;
    int total_input_num_in_blk = 0;
    CUDA_DEBUG_BLK(blk_tid, "(2) input bitmap num: %d, offset: %d\n", input_bitmap_num, input_bitmap_offset);
    // the block-length bitmap
    for(int i = 0; i < it; i++){
        int bitmap_idx = input_bitmap_offset+blk_tid;
        bitmap = input_bitmap[bitmap_idx];
        v_num[0] = __popc(bitmap);

        BlockScan(temp_storage).ExclusiveSum(v_num, v_offset, total_v_num_per_it);
        //__syncthreads();

        //Todo: binary search in bitmap
        int pos;
        for(int j = 0; j < v_num[0]; j++) {
            int input_offset = total_input_num_in_blk + v_offset[0] + j;
            pos = __ffs(bitmap) - 1;
            pos = bitmap_idx * 32 + pos;
            blk_input_frontiers[input_offset] = pos;
            bitmap &= bitmap - 1;
        }

        total_input_num_in_blk += total_v_num_per_it;
        input_bitmap_offset += block_size;
    }
    // the remainding bitmap
    if(blk_tid < blk_input_bitmap_remaind) {
        bitmap = input_bitmap[input_bitmap_offset+blk_tid];
        v_num[0] = __popc(bitmap);
        DEBUG_PRINT("(2) blk:%d, bitmap:0x%x, v_num:%d\n", blk_tid, bitmap, v_num[0]);
    } else {
        v_num[0] = 0;
    }

    BlockScan(temp_storage).ExclusiveSum(v_num, v_offset, total_v_num_per_it);
    //__syncthreads();
    //Todo: binary search for load balance
    int pos;
    for(int i = 0; i < v_num[0]; i++) {
        int input_offset = total_input_num_in_blk + v_offset[0] + i;
        pos = __ffs(bitmap) - 1;
        pos = (input_bitmap_offset + blk_tid) * 32 + pos;
        blk_input_frontiers[input_offset] = pos;
        bitmap &= bitmap - 1;
        DEBUG_PRINT("(2) input vertex id: %d\n", pos);
    }

    total_input_num_in_blk += total_v_num_per_it;
    CUDA_DEBUG_BLK(blk_tid, "(2) vertex num: %d\n", total_input_num_in_blk);

    //3.add vertex to update frontiers
    __shared__ int blk_update_offset;
    if (blk_tid == 0) {
        blk_update_offset = atomicAdd(&update_frontiers_num, total_input_num_in_blk);
        CUDA_DEBUG_BLK(blk_tid, "(3) update_num: %d, offset: %d\n", total_input_num_in_blk, blk_update_offset);
    }
    __syncthreads();

    for(int i = blk_tid; i < total_input_num_in_blk; i += block_size){
        update_frontiers[blk_update_offset+i] = blk_input_frontiers[i];
    }

    //4.travel and update blk_output_bitmap
    int it_time = total_input_num_in_blk / block_size;
    int blk_input_frontiers_remaind = total_input_num_in_blk % block_size;
    int neighbors_num;
    int vertex, start, end, neighbor;
    int total_ngbs;
    int index_in_bitmap, offset_in_bitmap;

    __shared__ int blk_degrees[BLOCK_MAX_SIZE];
    __shared__ int blk_start_offset[BLOCK_MAX_SIZE];
    //block
    for(int i = 0; i < it_time ; i++) {
        vertex = blk_input_frontiers[block_size*i+blk_tid];
        start = g_offset[vertex];
        blk_start_offset[blk_tid] = start;
        end = g_offset[vertex+1];
        neighbors_num = end - start;
        DEBUG_PRINT("(4) BLOCK:blk id: %d, vertex: %d, neighbor num: %d\n", blk_tid, vertex, neighbors_num);

        BlockScan(temp_storage).ExclusiveSum(neighbors_num, neighbors_num, total_ngbs);
        blk_degrees[blk_tid] = neighbors_num;
        __syncthreads();
        CUDA_DEBUG(glb_tid, "(4) BLOCK:total neighbors num: %d\n", total_ngbs);

        for(int j = blk_tid; j < total_ngbs; j += block_size){
            auto it = thrust::upper_bound(thrust::seq, blk_degrees, blk_degrees+block_size, j);
            int idx = thrust::distance(blk_degrees, it) - 1;

            vertex = blk_input_frontiers[block_size*i+idx];
            start = blk_start_offset[idx];
            int offset_in_ngb_per_vertex = j - blk_degrees[idx];
            neighbor = g_edges[start+offset_in_ngb_per_vertex];
            DEBUG_PRINT("(4) vertex: %d, neighbor: %d\n", vertex, neighbor);

            // mark output bitmap
            GET_BIT_INDEX_OFFSET(neighbor, index_in_bitmap, offset_in_bitmap);
            int bitmask = 1 << offset_in_bitmap;
            int thread_mask = __match_any_sync(__activemask(), index_in_bitmap);
            int first_tid = __ffs(thread_mask) - 1;
            int warp_or = __reduce_or_sync(thread_mask, bitmask);
            if(blk_tid % 32 == first_tid){
                bitmap = blk_status_bitmap[index_in_bitmap];
                bitmap ^= warp_or;
                warp_or &= bitmap;
                atomicOr(&blk_output_bitmap[index_in_bitmap], warp_or);
            }
        }
    }

    // the remainding vertex
    if(blk_tid < blk_input_frontiers_remaind){
        vertex = blk_input_frontiers[it_time*block_size+blk_tid];
        start = g_offset[vertex];
        blk_start_offset[blk_tid] = start;
        end = g_offset[vertex+1];
        neighbors_num = end - start;
        DEBUG_PRINT("(4) blk id: %d, vertex: %d, neighbor num: %d\n", blk_tid, vertex, neighbors_num);
    } else {
        neighbors_num = 0;
    }

    BlockScan(temp_storage).ExclusiveSum(neighbors_num, neighbors_num, total_ngbs);
    blk_degrees[blk_tid] = neighbors_num;
    __syncthreads();
    CUDA_DEBUG_BLK(blk_tid, "(4)total neighbors num: %d\n", total_ngbs);

    for(int i = blk_tid; i < total_ngbs; i += block_size){
        auto it = thrust::upper_bound(thrust::seq, blk_degrees, blk_degrees+blk_input_frontiers_remaind, i);
        int idx = thrust::distance(blk_degrees, it) - 1;

        vertex = blk_input_frontiers[block_size*it_time+idx];
        start = blk_start_offset[idx];
        int offset_in_ngb_per_vertex = i - blk_degrees[idx];
        neighbor = g_edges[start+offset_in_ngb_per_vertex];
        DEBUG_PRINT("(4) vertex: %d, neighbor: %d\n", vertex, neighbor);

        // mark output bitmap
        GET_BIT_INDEX_OFFSET(neighbor, index_in_bitmap, offset_in_bitmap);
        int bitmask = 1 << offset_in_bitmap;
        int thread_mask = __match_any_sync(__activemask(), index_in_bitmap);
        int first_tid = __ffs(thread_mask) - 1;
        int warp_or = __reduce_or_sync(thread_mask, bitmask);
        if(blk_tid % 32 == first_tid){
            bitmap = blk_status_bitmap[index_in_bitmap];
            bitmap ^= warp_or;
            warp_or &= bitmap;
            atomicOr(&blk_output_bitmap[index_in_bitmap], warp_or);
        }
    }
    __syncthreads();

    //5. aggregate the blk output bitmap to output bitmap
    it_time = bitmap_len / BLOCK_MAX_SIZE;
    int bitmap_rmd = bitmap_len % BLOCK_MAX_SIZE;
    int inserted_num;
    int pre_bitmap;
    int block_sum;
    for(int i = 0; i < it_time; i ++){
        bitmap = blk_output_bitmap[i*BLOCK_MAX_SIZE+blk_tid];
        pre_bitmap = bitmap == 0 ? 0 : atomicOr(&status_bitmap[i*BLOCK_MAX_SIZE+blk_tid], bitmap);
        DEBUG_PRINT("blk(%d, %d), pre_bitmap: %x, bitmap: %x\n", blockIdx.x, blk_tid, pre_bitmap, bitmap);

        pre_bitmap = bitmap == 0 ? 0 : atomicOr(&output_bitmap[i*BLOCK_MAX_SIZE+blk_tid], bitmap);
        pre_bitmap ^= bitmap;
        bitmap &= pre_bitmap;
        inserted_num = __popc(bitmap);
        block_sum = BlockReduce(temp_storage2).Sum(inserted_num);
        __syncthreads();
        if(blk_tid == 0) {
            blk_output_num += block_sum;
        }
        DEBUG_PRINT("blk(%d, %d) pre_bitmap: %x, bitmap: %x, inserted_num: %d, block_sum: %d\n", blockIdx.x, blk_tid, pre_bitmap, bitmap, inserted_num, block_sum);
    }

    if(blk_tid < bitmap_rmd){
        bitmap = blk_output_bitmap[it_time*BLOCK_MAX_SIZE+blk_tid];
        pre_bitmap = bitmap == 0 ? 0 : atomicOr(&status_bitmap[it_time*BLOCK_MAX_SIZE+blk_tid], bitmap);
        DEBUG_PRINT("rmd.blk(%d, %d), pre_bitmap: %x, bitmap: %x\n", blockIdx.x, blk_tid, pre_bitmap, bitmap);

        pre_bitmap = bitmap == 0 ? 0 : atomicOr(&output_bitmap[it_time*BLOCK_MAX_SIZE+blk_tid], bitmap);
        pre_bitmap ^= bitmap;
        bitmap &= pre_bitmap;
        inserted_num = __popc(bitmap);
    }
    else {
        inserted_num = 0;
    }

    block_sum = BlockReduce(temp_storage2).Sum(inserted_num);
    __syncthreads();
    if(blk_tid == 0) {
        blk_output_num += block_sum;
    }
    DEBUG_PRINT("rmd. blk(%d, %d) pre_bitmap: %x, bitmap: %x, inserted_num: %d, block_sum: %d\n", blockIdx.x, blk_tid, pre_bitmap, bitmap, inserted_num, block_sum);

    if(blk_tid == 0) {
        CUDA_DEBUG_BLK(blk_tid, "output_num: %d, blk_output_num: %d\n", *output_num, blk_output_num);
        atomicAdd(output_num, blk_output_num);

    }

}

__global__ void update_node_status_o2(int* input_bitmap, int offset, int nf_num, int* hops, int hop){
    int glb_tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(glb_tid < nf_num){
        int node = input_bitmap[offset+glb_tid];
        DEBUG_PRINT("offset: %d, node: %d, pre_hop: %d, hop: %d\n", offset+glb_tid, node, hops[node], hop);
        hops[node] = hop;
    }
}

void bfs_hops_async_o2(std::vector<int> offset, std::vector<int> edges, int node_num, int edge_num,
                   int source,
                   std::vector<int> &hops){
    int* d_offset, *d_edges;
    int offset_size = offset.size() * sizeof(int);
    int edge_size = edge_num * sizeof(int);

    int* d_input_bitmap; //int* d_nf_num;
    int* d_output_bitmap;
    int* d_status_bitmap;
    int word_bit_len = sizeof(int) * BYTE_SIZE;
    int bitmap_len = (node_num + word_bit_len - 1) / word_bit_len;
    int bitmap_size = bitmap_len * sizeof(int);
    int* bitmap = new int[bitmap_len];

    int* d_hops;
    int hops_size = node_num * sizeof(int);

    int* d_update_frontiers;
    int nf_size = node_num * sizeof(int);

    int* d_output_num; int output_num;
    int nf_num_size = sizeof(int);

    hipMalloc(&d_offset, offset_size);
    hipMalloc(&d_edges, edge_size);
    hipMalloc(&d_input_bitmap, bitmap_size);
    hipMalloc(&d_output_bitmap, bitmap_size);
    hipMalloc(&d_status_bitmap, bitmap_size);
    hipMalloc(&d_output_num, nf_num_size);
    hipMalloc(&d_hops, hops_size);
    hipMalloc(&d_update_frontiers, nf_size);
    // initial the input value
    hipMemcpy(d_offset, offset.data(), offset_size, hipMemcpyHostToDevice);
    hipMemcpy(d_edges, edges.data(), edge_size, hipMemcpyHostToDevice);

    int index_in_bitmap, offset_in_bitmap;
    GET_BIT_INDEX_OFFSET(source, index_in_bitmap, offset_in_bitmap);
    int bitmask = 1 << offset_in_bitmap;
    for(int i = 0; i < bitmap_len; i++){
        bitmap[i] = 0;
    }
    bitmap[index_in_bitmap] = bitmask;
    DEBUG_PRINT("initial input bitmap: index(%d), value(%d)\n", index_in_bitmap, bitmask);
    hipMemcpy(d_input_bitmap, bitmap, bitmap_size, hipMemcpyHostToDevice);
    hipMemcpy(d_status_bitmap, bitmap, bitmap_size, hipMemcpyHostToDevice);

    hipMemcpy(d_hops, hops.data(), hops_size, hipMemcpyHostToDevice);

    int initial_value = 0; int update_offset = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(update_frontiers_num), &initial_value, sizeof(int));

    hipStream_t stream_traversal, stream_update;
    hipStreamCreate(&stream_traversal);
    hipStreamCreate(&stream_update);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int sum = 0, level = 0;
    int threadsPerBlock, blocksPerGrid;
    int cur_hop = 0;
    int nf_num = 1;
    calculate_kernel_config(nf_num, threadsPerBlock, blocksPerGrid);
    hipEventRecord(start, 0);
    int threadsPerBlock_up = ((bitmap_len + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
    int threadsPerBlock_inital = bitmap_len > BLOCK_MAX_SIZE ? BLOCK_MAX_SIZE : threadsPerBlock_up;
    int blocksPerGrid_inital = (bitmap_len + BLOCK_MAX_SIZE - 1) / BLOCK_MAX_SIZE;

    while(nf_num && update_offset < 1500){
        int smem_size = (2 * bitmap_len + ((bitmap_len + blocksPerGrid - 1) / blocksPerGrid) * 32) * sizeof(int);
        //assert(smem_size < 48 * 1024);

        initial_output_bitmap_o2<<<blocksPerGrid_inital, threadsPerBlock_inital>>>(d_output_bitmap, d_output_num, bitmap_len);
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("Kernel launch 1 error: %s\n", hipGetErrorString(err));
        // }
        calculate_kernel_config(nf_num, threadsPerBlock, blocksPerGrid);
        get_output_frontiers_o2<<<blocksPerGrid, threadsPerBlock, smem_size>>>(d_offset, d_edges, node_num, edge_num,
                                                                                d_input_bitmap, nf_num,
                                                                                d_output_bitmap, d_output_num,
                                                                                d_status_bitmap, bitmap_len,
                                                                                d_update_frontiers);
        // err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("Kernel launch 2 error: %s\n", hipGetErrorString(err));
        // }

        hipMemcpy(&output_num, d_output_num, nf_num_size, hipMemcpyDeviceToHost);
        // err = hipDeviceSynchronize();
        // if (err != hipSuccess) {
        //     printf("CUDA error: %s\n", hipGetErrorString(err));
        //     //return;
        // }

        calculate_kernel_config(nf_num, threadsPerBlock, blocksPerGrid);
        update_node_status_o2<<<blocksPerGrid, threadsPerBlock, 0, stream_update>>>(d_update_frontiers, update_offset, nf_num,
                                                                   d_hops, cur_hop);
        update_offset += nf_num;
        cur_hop++;

        nf_num = output_num;
        DEBUG_PRINT("level: %d, output_num: %d\n", ++level, nf_num);
        int* tmp = d_input_bitmap;
        d_input_bitmap = d_output_bitmap;
        d_output_bitmap = tmp;

        //printf("update_offset: %d\n", update_offset);
    }
    //printf("sum: %d\n", sum);
    hipMemcpy(hops.data(), d_hops, hops_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU: bfs_hops_async_o2. Elapsed time :" << milliseconds << " (ms)\n";

    hipFree(d_offset);
    hipFree(d_edges);
    hipFree(d_hops);
    hipFree(d_input_bitmap);
    hipFree(d_output_bitmap);
    hipFree(d_status_bitmap);
    hipFree(d_output_num);
    hipFree(d_update_frontiers);

    hipStreamDestroy(stream_traversal);
    hipStreamDestroy(stream_update);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

std::vector<int> test_bfs_hops_async_o2(std::vector<int> offset, std::vector<int> endnodes, int source){
    int node_num = offset.size() - 1;
    int edge_num = endnodes.size();

    std::vector<int> hops(node_num, INVAILD);
    hops[source] = 0;
    // std::cout << "GPU:" << std::endl;

    bfs_hops_async_o2(offset, endnodes, node_num, edge_num, source, hops);

    //print_hops(source, hops);
    return hops;
}
