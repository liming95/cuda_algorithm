#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <cuda_pipeline.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
using namespace std;

void matrix_mul(float* A, float* B, float* C, int N){
    for(int i = 0; i < N; i++){
	for(int j = 0; j < N; j++){
	       C[i*N+j] = 0.0f;
	   for(int k = 0; k < N; k++){
	       C[i*N+j] += A[i*N+k] * B[k*N+j];
	   }
	}
    }
}

__global__ void matmul_kernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < N && col < N) {
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

//Tiled Matrix-Matrix Multiplication
__global__ void matmul_kernel_tiling(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float A_shared[TILE_DIM][TILE_DIM];
    __shared__ float B_shared[TILE_DIM][TILE_DIM];
    float sum = 0.0f;

    for(int i = 0; i < N / TILE_DIM; i++){
        A_shared[threadIdx.y][threadIdx.x] = A[row*N+i*TILE_DIM+threadIdx.x];
        B_shared[threadIdx.y][threadIdx.x] = B[(i*TILE_DIM+threadIdx.y)*N+col];
        __syncthreads();

        for(int j = 0; j < TILE_DIM; j++){
            sum += A_shared[threadIdx.y][j] * B_shared[j][threadIdx.x];
        }
    }

    // TODO: If N is not an integer multiple of TILE_DIM, the remaining part needs to be handled.
    C[row * N + col] = sum;
}

//Tiled and prefetch Matrix-Matrix Multiplication
__global__ void matmul_kernel_tiling_prefetch(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float A_shared[TILE_DIM][TILE_DIM];
    __shared__ float B_shared[TILE_DIM][TILE_DIM];
    float sum = 0.0f;
    float A_prefetch = A[row*N+threadIdx.x];
    float B_prefetch = B[threadIdx.y*N+col];
    for(int i = 0; i < N / TILE_DIM; i++){
        A_shared[threadIdx.y][threadIdx.x] = A_prefetch;
        B_shared[threadIdx.y][threadIdx.x] = B_prefetch;
        __syncthreads();

        if((i+1) < N/TILE_DIM){
            __pipeline_memcpy_async(&A_prefetch, &A[row*N+(i+1)*TILE_DIM+threadIdx.x], sizeof(float));
            __pipeline_memcpy_async(&B_prefetch, &B[((i+1)*TILE_DIM+threadIdx.y)*N+col], sizeof(float));
            __pipeline_commit();
            // A_prefetch = A[row*N+(i+1)*TILE_DIM+threadIdx.x];
            // B_prefetch = B[((i+1)*TILE_DIM+threadIdx.y)*N+col];
        }

        for(int j = 0; j < TILE_DIM; j++){
            sum += A_shared[threadIdx.y][j] * B_shared[j][threadIdx.x];
        }
        __pipeline_commit();
        __pipeline_wait_prior(0);
        //__syncthreads();
    }

    // TODO: If N is not an integer multiple of TILE_DIM, the remaining part needs to be handled.
    C[row * N + col] = sum;
}

//Tiled, prefetch, bank conflict Matrix-Matrix Multiplication
// shared memory conflict. degrate the performance. Maybe the reason is index calculation increase more latency than free conflict
// Todo: what is the type of situation where the random index for bank conflict is useful?
__global__ void matmul_kernel_tiling_prefetch_bank(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float A_shared[TILE_DIM][TILE_DIM];
    __shared__ float B_shared[TILE_DIM][TILE_DIM];
    float sum = 0.0f;
    float A_prefetch = A[row*N+threadIdx.x];
    float B_prefetch = B[threadIdx.y*N+col];
    for(int i = 0; i < N / TILE_DIM; i++){
        A_shared[threadIdx.y][threadIdx.x] = A_prefetch;
        B_shared[threadIdx.y][threadIdx.y ^ threadIdx.x] = B_prefetch;
        __syncthreads();

        if((i+1) < N/TILE_DIM){
            __pipeline_memcpy_async(&A_prefetch, &A[row*N+(i+1)*TILE_DIM+threadIdx.x], sizeof(float));
            __pipeline_memcpy_async(&B_prefetch, &B[((i+1)*TILE_DIM+threadIdx.y)*N+col], sizeof(float));
            __pipeline_commit();
        }

        for(int j = 0; j < TILE_DIM; j++){
            sum += A_shared[threadIdx.y][j] * B_shared[j][j^threadIdx.x];
        }
        __pipeline_commit();
        __pipeline_wait_prior(0);
    }

    // TODO: If N is not an integer multiple of TILE_DIM, the remaining part needs to be handled.
    C[row * N + col] = sum;
}
//Todo: register, hardware(tile size, block size), tensor core, device-host data transfer, collaberative.
void launch_matmul(float* d_A, float* d_B, float* d_C, int N) {
    dim3 threadsPerBlock(TILE_DIM, TILE_DIM);
    dim3 blocksPerGrid((N + TILE_DIM - 1) / TILE_DIM, (N + TILE_DIM - 1) / TILE_DIM);
    //matmul_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    //matmul_kernel_tiling<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    //matmul_kernel_tiling_prefetch<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matmul_kernel_tiling_prefetch_bank<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
}

void fill_arry(float* A, float* B, int N){
    for(int i = 0; i < N; i++){
        for (int j = 0; j < N; j++) {
            A[i*N+j] = static_cast<float>(rand() % 5);
            B[i*N+j] = static_cast<float>(rand() % 5);
        }
    }
}
void test_matrix_mul_cpu(){
    int row, col;
    row = MATRIX_SIZE;
    col = MATRIX_SIZE;
    float mat1[row][col], mat2[row][col];
    float result[row][col];
    fill_arry((float*)mat1, (float*)mat2, MATRIX_SIZE);

    auto start = std::chrono::high_resolution_clock::now();

    matrix_mul((float*)mat1, (float*)mat2, (float*)result, MATRIX_SIZE);

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    std::cout << "CPU matrix_mul " << duration.count() << " nanoseconds .\n";

}

void test_matrix_mul(){
    // initial matrix
    int row, col;
    row = MATRIX_SIZE;
    col = MATRIX_SIZE;
    float mat1[row][col], mat2[row][col];
    float result[row][col];
    fill_arry((float*)mat1, (float*)mat2, MATRIX_SIZE);
    matrix_mul((float*)mat1, (float*)mat2, (float*)result, MATRIX_SIZE);
    std::cout << "[CPU]:Result matrix C:\n";
    for (int i = 0; i < row; ++i) {
        //for (int j = 0; j < col; ++j) {
            std::cout << result[i][0] << "\t";
        //}
    }
    std::cout << "\n";

    // cpy from host to device
    int size = row * col * sizeof(float);
    float *d_mat1, *d_mat2, *d_result;

    auto start = std::chrono::high_resolution_clock::now();
    hipMalloc(&d_mat1, size);
    hipMalloc(&d_mat2, size);
    hipMalloc(&d_result, size);

    hipMemcpy(d_mat1, mat1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, mat2, size, hipMemcpyHostToDevice);

    // launch kernel
    launch_matmul(d_mat1, d_mat2, d_result, MATRIX_SIZE);

    // cpy result from device to host
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    std::cout << "GPU matrix_mul " << duration.count() << " nanoseconds .\n";
    // print result
    std::cout << "[GPU]:Result matrix C:\n";
    for (int i = 0; i < row; ++i) {
        //for (int j = 0; j < col; ++j) {
            std::cout << result[i][0] << "\t";
        //}
    }
    std::cout << "\n";
    // clean
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_result);
}
